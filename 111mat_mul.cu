#include<hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<math.h>
#include "hip/hip_runtime.h"
#include "hipsparse.h"
#include "hipblas.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
using namespace std;
extern "C++" void gpu_compute_sparse(int* h_csrRows, int* h_csrCols, float* h_csrVals, float* h_N, float* h_P, int C, int M, int K, int H, int W);

void gpu_compute_sparse(int* h_csrRows, int* h_csrCols, float* h_csrVals, float* h_N, float* h_P, int C, int M, int K, int H, int W){
    int nnz = h_csrRows[K * K * M];
    int out_col = H * W;

    int *d_csrRows,*d_csrCols;
    float alpha=1;
    float beta=0;
    float *d_csrVals;
    float *d_N, *d_P, *d_temp_P;

    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    checkCudaErrors(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
    checkCudaErrors(hipsparseCreate(&handle));

    checkCudaErrors(hipMalloc((void **)&d_csrVals, nnz*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_csrRows, (M+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrCols, nnz*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_N, out_col*C*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_P, out_col*(M)*sizeof(float)));

    hipMemcpy(d_csrVals,h_csrVals,nnz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csrRows,h_csrRows,(M+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrCols,h_csrCols,nnz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_N,h_N,out_col*C*sizeof(float), hipMemcpyHostToDevice);

    checkCudaErrors(hipMemset((void *)d_P,0, out_col*(M)*sizeof(float)));

    for(int i = 0;i < K*K; i++){
        checkCudaErrors(hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,M,out_col,C,nnz,&alpha,descr,d_csrVals,d_csrRows, d_csrCols,d_N,C,&beta,d_P,M));
    }
    checkCudaErrors(hipMemcpy(h_P,d_P,out_col*(M)*sizeof(float),hipMemcpyDeviceToHost));

    for (int i=0;i<M*out_col;i++)
    {
        cout<<h_P[i]<<" ";
    }

}