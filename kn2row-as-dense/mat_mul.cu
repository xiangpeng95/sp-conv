#include<hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<math.h>

#include "hipblas.h"
extern "C" void gpu_compute(float* h_M, float* h_N, float* h_P, int C, int M, int K, int H, int W);

__global__
void add_buffer_Kernel(float* d_P, float* d_temp_P, int M, int H, int W, int mark_row, int mark_col){
    int buffer_row, buffer_col, x, y;
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    if(Row < M && Col < H*W){
        y = Col / W;
        x = Col % W;

        buffer_row = y - mark_row;
        buffer_col = x - mark_col;

        if(buffer_row >= 0 && buffer_row < H && buffer_col >= 0 && buffer_col < W)
            d_P[H*W*Row + buffer_row * W + buffer_col] += d_temp_P[H*W*Row + y * W + x];
    }
}
void gpu_compute(float* h_M, float* h_N, float* h_P, int C, int M, int K, int H, int W) {

    float *d_M, *d_N, *d_P, *d_temp_P;

    size_t size_M = M * C * sizeof(float);
    size_t size_N = H * W * C * sizeof(float);
    size_t size_P = H * W * M * sizeof(float);


    hipMalloc((void**)&d_M, size_M);
    hipMalloc((void**)&d_N, size_N);
    hipMalloc((void**)&d_P, size_P);
    hipMalloc((void**)&d_temp_P, size_P);//充当buffer，记录中间值

    for(int i = 0;i < H * W * M; i++)h_P[i] = 0.0; //初始化

    hipMemcpy(d_P, h_P, size_P, hipMemcpyHostToDevice);//将数据输入GPU
    hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);//将数据输入GPU

    float alpha = 1;
    float beta = 0;
    float *h_temp_M = (float*)malloc(size_M);

    for(int i = 0;i < K*K; i++){
        memcpy(h_temp_M, h_M + i * M * C, M * C * sizeof(float));
        hipMemcpy(d_M, h_temp_M, size_M, hipMemcpyHostToDevice);

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        hipblasSgemm(handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            H*W,                  //矩阵N的列数
            M,                    //矩阵M的行数
            C,                    //矩阵M的列数
            &alpha,
            d_N,
            H*W,
            d_M,
            C,
            &beta,
            d_temp_P,
            H*W); //中间结果d_temp_P，偏移后加入d_P


        int mark_row, mark_col;
        mark_col = i % K;
        mark_row = i / K;
        mark_row -= K / 2;
        mark_col -= K / 2;

        dim3 dimGrid(ceil((H*W) / 16.0), ceil(M / 16.0), 1);
        dim3 dimBlock(16, 16, 1);

        add_buffer_Kernel<<<dimGrid, dimBlock>>>(d_P, d_temp_P, M, H, W, mark_row, mark_col);

    }


    hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);
    
    // Free device memory for M, N, P
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    hipFree(d_temp_P);

    free(h_temp_M);

}






















/*
for(int L = 0; L < M; L++) //一行一行的进行偏移，每一行的起始地址d_temp_P + H*W*l
            for(int y = 0; y < H; y++)
                for(int x = 0; x < W; x++){
                    row = y - mark_row;
                    col = x - mark_col;

                    if(row >= 0 && row < H && col >= 0 && col < W)
                        d_P[H*W*L + row * W + col] += d_temp_P[H*W*L + y * W + x];

                }
*/